
#include <hip/hip_runtime.h>
/*Tutorial followed: https://cuda-tutorial.readthedocs.io/en/latest/tutorials/tutorial01/
*/

__global__ void helloWorld(){ // __global__ specifies that this kernel(function) is to be run on device(GPU)
    printf("Hello World from GPU!\n");
}

// host code(CPU code)
int main() {
    helloWorld<<<1,1>>>(); // launch this kernel on just 1 thread of a block
    return 0;
}
