#include <iostream>
#include <hip/hip_runtime.h> // For CUDA functions and types

// __global__ means this is a kernel and will run on GPU
__global__ void saxpy(int n, float a, float *x, float *y) {

    // we parallelized the for loop
    int i = blockIdx.x * blockDim.x + threadIdx.x; // built in values for every single thread. Every thread knows where it is in a grid, then block. Compute global thread id
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

int main() {
    int n = 1000; // Example size of the vectors

     // Following the style used in many NVIDIA CUDA examples, we use the prefix h_ in naming pointer variables for memory allocated in CPU memory and d_ for pointers for memory allocated in GPU memory.
    float *h_x, *h_y; // Host memory
    float *d_x, *d_y; // Device memory

    // allocate host memory for h_x and h_y and initialize contents
    // cudaMalloc: invokes the GPU driver and asks it to allocate memory on the GPU for use by the program. 
    h_x = (float*) malloc(n * sizeof(float));
    h_y = (float*) malloc(n * sizeof(float));
    
    if (h_x == nullptr || h_y == nullptr) {
        std::cerr << "Host memory allocation failed!" << std::endl;
        return 1;
    }

    // Initialize host memory
    for (int i = 0; i < n; ++i) {
        h_x[i] = static_cast<float>(i);
        h_y[i] = static_cast<float>(i);
    }

    // Allocate device memory
    hipError_t err;
    err = hipMalloc(&d_x, n * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "Device memory allocation failed: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    err = hipMalloc(&d_y, n * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "Device memory allocation failed: " << hipGetErrorString(err) << std::endl;
        hipFree(d_x);
        return 1;
    }

    // Copy data from host to device
    err = hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Error copying data from host to device: " << hipGetErrorString(err) << std::endl;
        hipFree(d_x);
        hipFree(d_y);
        return 1;
    }
    err = hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Error copying data from host to device: " << hipGetErrorString(err) << std::endl;
        hipFree(d_x);
        hipFree(d_y);
        return 1;
    }

    int threadsPerBlock = 256; 
    int nblocks = (n + threadsPerBlock - 1) / threadsPerBlock; // Padding if the threads in block is less than desired(take round off for the block)
    saxpy<<<nblocks, threadsPerBlock>>>(n, 2.0f, d_x, d_y);

    // Check for any errors launching the kernel
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        hipFree(d_x);
        hipFree(d_y);
        return 1;
    }

    // Copy result from device to host
    err = hipMemcpy(h_y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Error copying data from device to host: " << hipGetErrorString(err) << std::endl;
        hipFree(d_x);
        hipFree(d_y);
        return 1;
    }

    // Print the result for verification, comment if doing time analysis
    for (int i = 0; i < n; ++i) {
        std::cout << "h_y[" << i << "] = " << h_y[i] << std::endl;
    }

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);

    // Free host memory
    free(h_x);
    free(h_y);

    return 0;
}
